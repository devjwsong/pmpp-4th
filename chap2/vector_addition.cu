
#include <hip/hip_runtime.h>
/*
Compute vector sum C = A + B
Each thread preforms one pair-wise addition
*/
__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x  + blockDim.x * blockIdx.x;
    if (n < 100) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    // Allocate the device global memory for each array.
    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    // Copy the values in A and B from host to device.
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Execute the kernel function.
    vecAddKernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    // Copy the values in C from device to host.
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Free the allocated memory spaces in the device.
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
